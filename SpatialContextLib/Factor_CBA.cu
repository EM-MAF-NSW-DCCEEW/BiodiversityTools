#include "hip/hip_runtime.h"
/*
Context_CBA.cpp - CUDA CBA functions for performing BFT context analysis with factor grid
Copyright(C) 2024 State of New South Wales and Department of Climate Change, Energy, the Environment and Water (DCCEEW)
Author: Jamie Love, Ecological Modelling, Science and Insights Division

This program is free software : you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.If not, see <https://www.gnu.org/licenses/>.
*/


#ifndef _FACTOR_CBA_KERNEL_CU_
#define _FACTOR_CBA_KERNEL_CU_
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <memory>
#include "Common.cuh"
#include "Parameters.h"
#include "FileUtilsLib.h"

#ifndef USE_PRECALC_EDGE_KERNELS
__global__ void FactorCBA_kernel(int inOffset, int outOffset, int2 *d_petalData, float *d_outData, hipTextureObject_t tex4Obj)
{
	//Focal cell
	float4 focal = tex2D<float4>(tex4Obj, blockIdx.x, blockIdx.y + inOffset);
	if ((focal.x == 0.0f && d_multFocal) || focal.y == 0.0f)
		d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = 0.0f;
	else if (focal.x == d_noData || focal.y == d_noData || focal.z == d_noData)
		d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = d_noData;
	else {
		//Declare thread local storage
		int         i;
		int2 intPairs[4];
		float4 t;

		//Address shared memory
		extern __shared__ float pBenefit[];         //habitat
		float *pCost = &pBenefit[blockDim.x];       //permeability
		float *pDist = &pBenefit[2 * blockDim.x];   //accum cost
		float *pCount = &pBenefit[3 * blockDim.x];  //count row
		pBenefit[threadIdx.x] = 0.0f;
		pCost[threadIdx.x] = 0.0f;
		pDist[threadIdx.x] = 0.0f;
		pCount[threadIdx.x] = 0.0f;

		//Aggregate cell values into petals
		intPairs[0] = d_petalData[4 * blockDim.x + threadIdx.x];
		intPairs[0].y = 0;
		for (i = 0; i < intPairs[0].x; i++) {
			intPairs[1] = d_petalData[(i + 5) * blockDim.x + threadIdx.x];
			if (intPairs[1].x != -1 || intPairs[1].y != -1) {
				t = tex2D<float4>(tex4Obj, float(int(blockIdx.x) + intPairs[1].x - d_focalOffset), float(inOffset + int(blockIdx.y) + intPairs[1].y - d_focalOffset));
				if (t.x >= 0.0f && t.y > 0.0f && t.z >= 0.0f) {
					pBenefit[threadIdx.x] += (t.x * t.z);
					pCost[threadIdx.x] += t.y;
					++(intPairs[0].y);
				}
			}
		}

		//Normalise petal cost and set petal counter
		if (intPairs[0].y > 0) {
			pCost[threadIdx.x] = powf(pCost[threadIdx.x] / (float)intPairs[0].y, 2.0f * sqrtf((float)intPairs[0].y * d_oneOnPi));
			pCount[threadIdx.x] = 1.0f;
		}

		//Get petal neighbour indices N, S, W, E, NW, NE, SW, SE
		intPairs[0] = d_petalData[0 * blockDim.x + threadIdx.x];
		intPairs[1] = d_petalData[1 * blockDim.x + threadIdx.x];
		intPairs[2] = d_petalData[2 * blockDim.x + threadIdx.x];
		intPairs[3] = d_petalData[3 * blockDim.x + threadIdx.x];

		//Calculate permeability for eight focal cell neighbours first
		if (threadIdx.x < 8)
			pDist[threadIdx.x] = threadIdx.x % 2 == 0 ? powf(pCost[threadIdx.x] * focal.y, d_diag) : sqrtf(pCost[threadIdx.x] * focal.y);

		//Loop until all permeabilities are maximised (i==0)
		i = 1;
		while (__syncthreads_or(i)) {
			i = 0;
			if (intPairs[0].x != -1 && pDist[intPairs[0].x] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[0].x]) > pDist[threadIdx.x]) {//N                
				pDist[threadIdx.x] = pDist[intPairs[0].x] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[0].x]);
				i++;
			}
			if (intPairs[0].y != -1 && pDist[intPairs[0].y] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[0].y]) > pDist[threadIdx.x]) {//S                
				pDist[threadIdx.x] = pDist[intPairs[0].y] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[0].y]);
				i++;
			}
			if (intPairs[1].x != -1 && pDist[intPairs[1].x] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[1].x]) > pDist[threadIdx.x]) {//E                
				pDist[threadIdx.x] = pDist[intPairs[1].x] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[1].x]);
				i++;
			}
			if (intPairs[1].y != -1 && pDist[intPairs[1].y] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[1].y]) > pDist[threadIdx.x]) {//W                
				pDist[threadIdx.x] = pDist[intPairs[1].y] * sqrtf(pCost[threadIdx.x] * pCost[intPairs[1].y]);
				i++;
			}
			if (intPairs[2].x != -1 && pDist[intPairs[2].x] * powf(pCost[threadIdx.x] * pCost[intPairs[2].x], d_diag) > pDist[threadIdx.x]) {//NW                
				pDist[threadIdx.x] = pDist[intPairs[2].x] * powf(pCost[threadIdx.x] * pCost[intPairs[2].x], d_diag);
				i++;
			}
			if (intPairs[2].y != -1 && pDist[intPairs[2].y] * powf(pCost[threadIdx.x] * pCost[intPairs[2].y], d_diag) > pDist[threadIdx.x]) {//NE               
				pDist[threadIdx.x] = pDist[intPairs[2].y] * powf(pCost[threadIdx.x] * pCost[intPairs[2].y], d_diag);
				i++;
			}
			if (intPairs[3].x != -1 && pDist[intPairs[3].x] * powf(pCost[threadIdx.x] * pCost[intPairs[3].x], d_diag) > pDist[threadIdx.x]) {//SW               
				pDist[threadIdx.x] = pDist[intPairs[3].x] * powf(pCost[threadIdx.x] * pCost[intPairs[3].x], d_diag);
				i++;
			}
			if (intPairs[3].y != -1 && pDist[intPairs[3].y] * powf(pCost[threadIdx.x] * pCost[intPairs[3].y], d_diag) > pDist[threadIdx.x]) {//SE               
				pDist[threadIdx.x] = pDist[intPairs[3].y] * powf(pCost[threadIdx.x] * pCost[intPairs[3].y], d_diag);
				i++;
			}
		}

		//Calculate HjWij
		pDist[threadIdx.x] *= pBenefit[threadIdx.x];

		//Sum all HjWij and petal count
		__syncthreads();
		for (i = d_firstReduction; i > 0; i /= 2) {
			if (threadIdx.x < i && threadIdx.x + i < d_nPetals) {
				pDist[threadIdx.x] += pDist[threadIdx.x + i];
				pCount[threadIdx.x] += pCount[threadIdx.x + i];
			}
			__syncthreads();
		}

		//Final focal cell calculations then write NHA to outData
		if (threadIdx.x == 0) {
			pDist[0] += (focal.x * focal.z);
			pDist[0] *= (pCount[0] < (float)d_nPetals) ? ((float)d_nPetals) / (pCount[0] + 1.0f) : 1.0f;
			pDist[0] *= d_multFocal ? powf(focal.x, d_focalPower) : 1.0f;
			d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = powf(pDist[0], d_sumPower);
		}
	}
}

#else
//NOTE Do NOT use causes non - reproducable errors
__global__ void FactorCBA_kernel(int inOffset, int outOffset, int2 *d_petalData, float *d_outData)
{
	//Get an check the focal cell
	float4 focal = tex2D(tex4Ref, blockIdx.x, blockIdx.y + inOffset);
	if (focal.y == 0.0f)
		d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = 0.0f;
	else if (focal.x == d_noData || focal.y == d_noData || focal.z == d_noData)
		d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = d_noData;
	else {

		//Declare thread local storage
		int i;
		float Hj = 0.0f;
		int2 intPairs[4];
		float4 costNSWE{ -1.0f, -1.0f, -1.0f, -1.0f }; //also used for fetching cell values
		float4 costNDSD{ -1.0f, -1.0f, -1.0f, -1.0f };

		//Address shared memory
		extern __shared__ float pDist[];
		int *pCount = (int*)&pDist[blockDim.x];
		pDist[threadIdx.x] = 0.0f; //Cost, Distance
		pCount[threadIdx.x] = 0; //Active, Count

								 //Calculate petal values from cells
		intPairs[0] = d_petalData[4 * blockDim.x + threadIdx.x];
		intPairs[0].y = 0;
		for (i = 0; i < intPairs[0].x; i++) {
			intPairs[1] = d_petalData[(i + 5) * blockDim.x + threadIdx.x];
			if (intPairs[1].x != -1 || intPairs[1].y != -1) {
				costNSWE = tex2D(tex4Ref, float(int(blockIdx.x) + intPairs[1].x - d_focalOffset), float(inOffset + int(blockIdx.y) + intPairs[1].y - d_focalOffset));
				if (costNSWE.y > d_fltEps) {
					Hj += (costNSWE.x * costNSWE.z);
					pDist[threadIdx.x] += costNSWE.y;
					++(intPairs[0].y);
				}
			}
		}

		//Normalise petal cost and set petal counter
		if (intPairs[0].y > 0) {
			pDist[threadIdx.x] = powf(pDist[threadIdx.x] / (float)intPairs[0].y, 2.0f * sqrtf((float)intPairs[0].y * d_oneOnPi));
			pCount[threadIdx.x] = 1;
		}

		//Get petal neighbour indices
		intPairs[0] = d_petalData[0 * blockDim.x + threadIdx.x];//N, S
		intPairs[1] = d_petalData[1 * blockDim.x + threadIdx.x];//W, E
		intPairs[2] = d_petalData[2 * blockDim.x + threadIdx.x];//NW, NE
		intPairs[3] = d_petalData[3 * blockDim.x + threadIdx.x];//SW, SE

															   //Calculate petal edge costs
		__syncthreads();
		costNSWE.x = intPairs[0].x != -1 ? sqrtf(pDist[threadIdx.x] * pDist[intPairs[0].x]) : -1.0f;
		costNSWE.y = intPairs[0].y != -1 ? sqrtf(pDist[threadIdx.x] * pDist[intPairs[0].y]) : -1.0f;
		costNSWE.z = intPairs[1].x != -1 ? sqrtf(pDist[threadIdx.x] * pDist[intPairs[1].x]) : -1.0f;
		costNSWE.w = intPairs[1].y != -1 ? sqrtf(pDist[threadIdx.x] * pDist[intPairs[1].y]) : -1.0f;
		costNDSD.x = intPairs[2].x != -1 ? powf(pDist[threadIdx.x] * pDist[intPairs[2].x], d_diag) : -1.0f;
		costNDSD.y = intPairs[2].y != -1 ? powf(pDist[threadIdx.x] * pDist[intPairs[2].y], d_diag) : -1.0f;
		costNDSD.z = intPairs[3].x != -1 ? powf(pDist[threadIdx.x] * pDist[intPairs[3].x], d_diag) : -1.0f;
		costNDSD.w = intPairs[3].y != -1 ? powf(pDist[threadIdx.x] * pDist[intPairs[3].y], d_diag) : -1.0f;

		//Calculate permeability for focal cell neighbours first
		if (threadIdx.x < 8)
			pDist[threadIdx.x] = threadIdx.x % 2 == 0 ? powf(pDist[threadIdx.x] * focal.y, d_diag) : sqrtf(pDist[threadIdx.x] * focal.y);
		else
			pDist[threadIdx.x] = 0.0f;

		//Loop until all permeabilities are maximised (i==0) then multiply by Hj 
		i = 1;
		while (__syncthreads_or(i)) {
			i = 0;
			if (intPairs[0].x != -1 && pDist[intPairs[0].x] * costNSWE.x > pDist[threadIdx.x]) {//N edge
				pDist[threadIdx.x] = pDist[intPairs[0].x] * costNSWE.x;
				i = 1;
			}
			if (intPairs[0].y != -1 && pDist[intPairs[0].y] * costNSWE.y > pDist[threadIdx.x]) {//S edge
				pDist[threadIdx.x] = pDist[intPairs[0].y] * costNSWE.y;
				i = 1;
			}
			if (intPairs[1].x != -1 && pDist[intPairs[1].x] * costNSWE.z > pDist[threadIdx.x]) {//W edge
				pDist[threadIdx.x] = pDist[intPairs[1].x] * costNSWE.z;
				i = 1;
			}
			if (intPairs[1].y != -1 && pDist[intPairs[1].y] * costNSWE.w > pDist[threadIdx.x]) {//E edge
				pDist[threadIdx.x] = pDist[intPairs[1].y] * costNSWE.w;
				i = 1;
			}
			if (intPairs[2].x != -1 && pDist[intPairs[2].x] * costNDSD.x > pDist[threadIdx.x]) {//NW edge
				pDist[threadIdx.x] = pDist[intPairs[2].x] * costNDSD.x;
				i = 1;
			}
			if (intPairs[2].y != -1 && pDist[intPairs[2].y] * costNDSD.y > pDist[threadIdx.x]) {//NE edge
				pDist[threadIdx.x] = pDist[intPairs[2].y] * costNDSD.y;
				i = 1;
			}
			if (intPairs[3].x != -1 && pDist[intPairs[3].x] * costNDSD.z > pDist[threadIdx.x]) {//SW edge
				pDist[threadIdx.x] = pDist[intPairs[3].x] * costNDSD.z;
				i = 1;
			}
			if (intPairs[3].y != -1 && pDist[intPairs[3].y] * costNDSD.w > pDist[threadIdx.x]) {//SE edge
				pDist[threadIdx.x] = pDist[intPairs[3].y] * costNDSD.w;
				i = 1;
			}
		}
		pDist[threadIdx.x] *= Hj;

		//Sum all HjWij and petal count
		__syncthreads();
		for (i = d_firstReduction; i > 0; i /= 2) {
			if (threadIdx.x < i && threadIdx.x + i < d_nPetals) {
				pDist[threadIdx.x] += pDist[threadIdx.x + i];
				pCount[threadIdx.x] += pCount[threadIdx.x + i];
			}
			__syncthreads();
		}

		//Final focal cell calcs then write result to outData
		if (threadIdx.x == 0) {
			pDist[0] += (focal.x * focal.z);
			if (pCount[0] < d_nPetals) pDist[0] *= float(d_nPetals) / (float(pCount[0]) + 1.0f);
			pDist[0] *= d_multFocal ? powf(focal.x, d_focalPower) : 1.0f;
			d_outData[(outOffset + blockIdx.y) * gridDim.x + blockIdx.x] = powf(pDist[0], d_sumPower);
		}
	}
}

#endif

////////////////////////////////////////////////////////////
//Context CBA small
////////////////////////////////////////////////////////////
int CUDAFactorCBA_S(CBAParams &p)
{
	hipError_t cudaStatus;
	msgText("Performing Factor CBA");

	//Host and Device data
	float4 *h_inData, *d_inData;
	float *h_outData, *d_outData;
	int2 *d_petalData;

	size_t d_pitch;
	dim3 gridSize{ (unsigned int)p.nCols, 1U, 1U };
	uint maxKernelTime = h_maxKernelTime; // 100000;
	uint kernelTime = maxKernelTime;
	uint i;
	
	//Host 5 x 4bytes x nCells, Device 5 x 4bytes x nCells
	//Malloc pinned host memory
	msgText("Allocating host memory");
	CUDA(hipHostMalloc(&h_inData, p.nCells * sizeof(float4)));
	CUDA(hipHostMalloc(&h_outData, p.nCells * sizeof(float)));

	//Malloc device data
	msgText("Allocating device memory");
	CUDA(hipMalloc(&d_petalData, sizeof(int2) *p. petalRows * p.petalCols));
	CUDA(hipMallocPitch(&d_inData, &d_pitch, p.nCols * sizeof(float4), p.nRows));
	CUDA(hipMalloc(&d_outData, p.nCells * sizeof(float)));

	//Read h_inData from disk
	msgText("Reading input data to host memory");
	std::unique_ptr<float[]> inData = std::make_unique<float[]>(p.nCells);
	p.habInFS.read((char*)inData.get(), p.nCells * sizeof(float));
	for (i = 0; i < p.nCells; i++) { h_inData[i].x = inData[i]; }
	p.prmInFS.read((char*)inData.get(), p.nCells * sizeof(float));
	for (i = 0; i < p.nCells; i++) { h_inData[i].y = inData[i]; }
	p.facInFS.read((char*)inData.get(), p.nCells * sizeof(float));
	for (i = 0; i < p.nCells; i++) { h_inData[i].z = inData[i]; }
	inData.reset();

	//Move data to device
	msgText("Moving input data to device memory");
	CUDA(hipMemcpy(d_petalData, p.petalPtr.get(), sizeof(int2) * p.petalRows * p.petalCols, hipMemcpyHostToDevice));
	CUDA(hipMemcpy2D(d_inData, d_pitch, h_inData, p.nCols * sizeof(float4), p.nCols * sizeof(float4), p.nRows, hipMemcpyHostToDevice));
	CUDA(hipHostFree(h_inData));

	//Copy constants to device
	msgText("Setting device parameters");
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_nPetals), &(p.nPetals), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_focalOffset), &(p.fOffset), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_multFocal), &(p.multFocal), sizeof(bool)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_firstReduction), &(p.firstReduction), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_sumPower), &(p.sumPower), sizeof(float)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_focalPower), &(p.focalPower), sizeof(float)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_noData), &(p.noData), sizeof(float)));

	//Texture Reference
	//msgText("Setting texture reference");
	//tex4Ref.filterMode     = hipFilterModePoint;
	//tex4Ref.addressMode[0] = hipAddressModeBorder;
	//tex4Ref.addressMode[1] = hipAddressModeBorder;
	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	//CUDA(hipBindTexture2D (NULL, &tex4Ref, d_inData, &channelDesc, p.nCols, p.nRows, d_pitch));

	//Replace Texture Reference with Texture Object
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
	resDesc.res.pitch2D.devPtr = d_inData;
	resDesc.res.pitch2D.width = p.nCols;
	resDesc.res.pitch2D.height = p.nRows;
	resDesc.res.pitch2D.pitchInBytes = d_pitch;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = hipFilterModePoint;
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex4Obj = 0;
	CUDA(hipCreateTextureObject(&tex4Obj, &resDesc, &texDesc, NULL));

	//Profiled Kernel Call Loop
	msgText("Processing data on device");
	Profiler profiler(1000000), kernelTimer(1000000);
	profiler.Start();
	for (i = 0; i < p.nRows; i += gridSize.y) {
		//Set gridSize.y
		gridSize.y = max(gridSize.y * maxKernelTime / kernelTime, 1);
		if (i + gridSize.y >= p.nRows) gridSize.y = p.nRows - i;

		kernelTimer.Start();
		FactorCBA_kernel <<<gridSize, p.petalCols, p.petalCols * 4 * sizeof(float) >>>(i, i, d_petalData, d_outData, tex4Obj);
		CUDA(hipDeviceSynchronize());
		kernelTime = int(kernelTimer.Stop());
		msgProgress("Percent complete: ", i * 100 / p.nRows);
	}
	profiler.Stop();
	msgText("\rPercent complete: 100");
	msgText(("Processing time: " + toStr(profiler.Total())).c_str());

	//Copy device output data to host and write to disk
	msgText("Writing output data to disk");
	CUDA(hipMemcpy(h_outData, d_outData, p.nCells * sizeof(float), hipMemcpyDeviceToHost));
	p.cxtOutFS.write((const char*)h_outData, p.nCells * sizeof(float));

	//Free pinned Host Memory
	msgText("Freeing host and device memory");
	CUDA(hipDestroyTextureObject(tex4Obj));
	CUDA(hipHostFree(h_outData));
	CUDA(hipFree(d_petalData));
	CUDA(hipFree(d_inData));
	CUDA(hipFree(d_outData));

	cudaStatus = hipGetLastError();
	CUDA(hipDeviceReset());
	msgText((std::string("Device status ") + hipGetErrorString(cudaStatus)).c_str());
	msgText("CUDAFactor_CBA_S() Complete!");
	return 0;
}

////////////////////////////////////////////////////////////
//Context CBA large
////////////////////////////////////////////////////////////
int CUDAFactorCBA_L(CBAParams &p)
{
	hipError_t cudaStatus;
	msgText("Performing Factor CBA");

	//Host and Device data and buffers
	float4 *h_inData, *d_inData;
	float4 *h_inBuf, *d_inBuf;
	float *h_outBuf, *d_outBuf;
	int2 *d_petalData;

	size_t d_pitch;
	uint nBufferRows = p.fOffset * 2;
	uint nDataRows = p.fOffset * 4;
	uint nFirstRows = nDataRows - p.fOffset;
	uint nReads = 0;
	dim3 gridSize((unsigned int)p.nCols, 1U, 1U);
	uint maxKernelTime = h_maxKernelTime;
	uint kernelTime = maxKernelTime;
	uint i, j;

	//Malloc pinned host memory
	msgText("Allocating host memory");
	CUDA(hipHostMalloc(&h_inData, p.nCols * nDataRows * sizeof(float4)));
	CUDA(hipHostMalloc(&h_inBuf, p.nCols * nBufferRows * sizeof(float4)));
	CUDA(hipHostMalloc(&h_outBuf, p.nCols * nBufferRows * sizeof(float)));

	//Malloc device data
	msgText("Allocating device memory");
	CUDA(hipMalloc(&d_petalData, sizeof(int2) * p.petalRows * p.petalCols));
	CUDA(hipMallocPitch(&d_inData, &d_pitch, p.nCols * sizeof(float4), nDataRows));
	CUDA(hipMallocPitch(&d_inBuf, &d_pitch, p.nCols * sizeof(float4), nBufferRows));
	CUDA(hipMalloc(&d_outBuf, p.nCols * nBufferRows * sizeof(float)));

	//Read h_inData from disk
	msgText("Reading input data to host memory");
	for (i = 0; i < nDataRows * p.nCols; i++) {
		h_inData[i].x = 0;
		h_inData[i].y = 0;
		h_inData[i].z = 0;
		if (i >= p.fOffset * p.nCols) {
			p.habInFS.read((char*)&(h_inData[i].x), sizeof(float));
			p.prmInFS.read((char*)&(h_inData[i].y), sizeof(float));
			p.facInFS.read((char*)&(h_inData[i].z), sizeof(float));
		}
	}

	//Copy data to device
	msgText("Moving input data to device memory");
	CUDA(hipMemcpy(d_petalData, p.petalPtr.get(), sizeof(int2) * p.petalRows * p.petalCols, hipMemcpyHostToDevice));
	CUDA(hipMemcpy2D(d_inData, d_pitch, h_inData, p.nCols * sizeof(float4), p.nCols * sizeof(float4), nDataRows, hipMemcpyHostToDevice));
	
	//Copy constants to device
	msgText("Setting device parameters");
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_nPetals), &(p.nPetals), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_focalOffset), &(p.fOffset), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_multFocal), &(p.multFocal), sizeof(bool)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_firstReduction), &(p.firstReduction), sizeof(int)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_sumPower), &(p.sumPower), sizeof(float)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_focalPower), &(p.focalPower), sizeof(float)));
	CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_noData), &(p.noData), sizeof(float)));

	//Init Texture Reference
	//msgText("Setting texture reference");
	//tex4Ref.filterMode     = hipFilterModePoint;
	//tex4Ref.addressMode[0] = hipAddressModeBorder;
	//tex4Ref.addressMode[1] = hipAddressModeBorder;
	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	//CUDA(hipBindTexture2D (NULL, &tex4Ref, d_inData, &channelDesc, p.nCols, nDataRows, d_pitch));
	
	//Replace Texture Reference with Texture Object
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
	resDesc.res.pitch2D.devPtr = d_inData;
	resDesc.res.pitch2D.width = p.nCols;
	resDesc.res.pitch2D.height = nDataRows;
	resDesc.res.pitch2D.pitchInBytes = d_pitch;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = hipFilterModePoint;
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex4Obj = 0;
	CUDA(hipCreateTextureObject(&tex4Obj, &resDesc, &texDesc, NULL));

	//Profiled Kernel Call Loop
	msgText("Processing data on device");
	Profiler profiler(1000000), kernelTimer(1000000);
	profiler.Start();
	for (i = 0; i < p.nRows; i += gridSize.y) {
		//Set gridSize.y
		gridSize.y = max(min(gridSize.y * maxKernelTime / kernelTime, nBufferRows), 1);
		if (i + gridSize.y >= p.nRows) gridSize.y = p.nRows - i;

		//Parallel CBA kernel call
		kernelTimer.Start();
		FactorCBA_kernel <<<gridSize, p.petalCols, p.petalCols * 4 * sizeof(float) >>>(p.fOffset, 0, d_petalData, d_outBuf, tex4Obj);
		CUDA(hipDeviceSynchronize());
		kernelTime = (uint)kernelTimer.Stop();
		msgProgress("Percent complete: ", i * 100 / p.nRows);

		//Clear gridSize.y rows in the host buffer then read in any new data
		nReads = (nFirstRows + i) * p.nCols;
		for (j = 0; j < p.nCols * gridSize.y; j++) {
			h_inBuf[j].x = 0.0f;
			h_inBuf[j].y = 0.0f;
			h_inBuf[j].z = 0.0f;
			if (nReads + j < p.nCells) {
				p.habInFS.read((char*)&(h_inBuf[j].x), sizeof(float));
				p.prmInFS.read((char*)&(h_inBuf[j].y), sizeof(float));
				p.facInFS.read((char*)&(h_inBuf[j].z), sizeof(float));
			}
		}
		//Copy host buffer to device buffer then shuffle into inData
		CUDA(hipMemcpy2D(d_inBuf, d_pitch, h_inBuf, p.nCols * sizeof(float4), p.nCols * sizeof(float4), gridSize.y, hipMemcpyHostToDevice));
		ShuffleUp(d_inData, d_inBuf, nDataRows, gridSize.y, int(d_pitch / sizeof(float4)), true);

		//Move device output buffer to host then write to disk
		CUDA(hipMemcpy(h_outBuf, d_outBuf, p.nCols * gridSize.y * sizeof(float), hipMemcpyDeviceToHost));
		CUDA(hipMemset(d_outBuf, 0, p.nCols * gridSize.y * sizeof(float)));
		p.cxtOutFS.write((const char*)h_outBuf, p.nCols * gridSize.y * sizeof(float));

		CUDA(hipDeviceSynchronize());
	}	

	profiler.Stop();
	msgText("\rPercent complete: 100");
	msgText(("Processing time: " + toStr(profiler.Total())).c_str());

	//Free pinned Host Memory
	msgText("Freeing host and device memory");
	CUDA(hipDestroyTextureObject(tex4Obj));
	CUDA(hipHostFree(h_inData));
	CUDA(hipHostFree(h_inBuf));
	CUDA(hipHostFree(h_outBuf));
	CUDA(hipFree(d_petalData));
	CUDA(hipFree(d_inData));
	CUDA(hipFree(d_inBuf));
	CUDA(hipFree(d_outBuf));

	cudaStatus = hipGetLastError();
	CUDA(hipDeviceReset());
	msgText((std::string("Device status ") + hipGetErrorString(cudaStatus)).c_str());
	msgText("CUDAFactor_CBA_L() Complete!");
	return int(cudaStatus);
}


/**/
#endif